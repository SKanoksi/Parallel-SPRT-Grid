#include "hip/hip_runtime.h"
/*************************************
    Parallel Shortest Path Solver
             developed by
 Somrath Kanoksirirath
 Fourth year physics student,
 Faculty of Science,
 Mahidol University, Thailand.
*************************************/

#define DIM 3
#define TyPe float

#include "PSPS_CUDA_ExtendedBF_3D_Solver.h"

PSPS_CUDA_ExtendedBF_3D_Solver::PSPS_CUDA_ExtendedBF_3D_Solver()
{

}

PSPS_CUDA_ExtendedBF_3D_Solver::~PSPS_CUDA_ExtendedBF_3D_Solver()
{

}

//**************** CUDA program ****************//

__constant__ int shared_X ;
__constant__ int shared_Y ;
__constant__ int shared_Z ;
__constant__ int group_X  ;
__constant__ int group_Y  ;
__constant__ int group_Z  ;
__constant__ int radius_X ;
__constant__ int radius_Y ;
__constant__ int radius_Z ;

surface<void, cudaSurfaceType3D>  SlownessMap, Traveltime_A, Traveltime_B ;
surface<void, cudaSurfaceType3D>  UpdateMap_A,  UpdateMap_B, RaypathMap ;

// Calculate Weight (Slowness*LineWeight): common function
__device__ float SPR(const float *Slowness, const int ix, const int iy, const int iz, const int3 Source) // Vertex = relative directional vector wrt. source(posOnShmem)
{
	      int3    nVec   = make_int3( abs(ix), abs(iy), abs(iz) ) ;
	const float3  fVec   = make_float3(ix, iy, iz) ;
	const float3  stride = make_float3( 1.0f/nVec.x, 1.0f/nVec.y, 1.0f/nVec.z) ;
	
    float   Weight = 0 ;
	float3  cross  = make_float3( 1.0f-0.5f*stride.x, 1.0f-0.5f*stride.y, 1.0f-0.5f*stride.z) ;
	float crossOld = 1.0f ;


	// Main process
	while( 0 < nVec.x || 0 < nVec.y || 0 < nVec.z )
	{
        int next = 2 ;
        if( cross.y<cross.x ){            
            if( cross.z<cross.x ){ next=0; }
        }else{
            if( cross.z<cross.y ){ next=1; }
        }

		// Find the next largest crossing point wrt. crossOld	 
		if( next==0 ){

			// Compute weight.
			int indexX = Source.x + __float2int_rn( 0.5f*(crossOld+cross.x)*fVec.x ) ;
			int indexY = Source.y + __float2int_rn( 0.5f*(crossOld+cross.x)*fVec.y ) ;
			int indexZ = Source.z + __float2int_rn( 0.5f*(crossOld+cross.x)*fVec.z ) ;
			Weight += Slowness[ (indexZ*shared_Y + indexY)*shared_X + indexX ]*(crossOld-cross.x) ;

			// For next crossing point.
			crossOld = cross.x ;
			--nVec.x ;
			cross.x -= stride.x ;
		
		}else{
		if( next==1 ){
				
			// Compute weight.
			int indexX = Source.x + __float2int_rn( 0.5f*(crossOld+cross.y)*fVec.x ) ;
			int indexY = Source.y + __float2int_rn( 0.5f*(crossOld+cross.y)*fVec.y ) ;
			int indexZ = Source.z + __float2int_rn( 0.5f*(crossOld+cross.y)*fVec.z ) ;
			Weight += Slowness[ (indexZ*shared_Y + indexY)*shared_X + indexX ]*(crossOld-cross.y) ;

			// For next crossing point.
			crossOld = cross.y ;
			--nVec.y ;
			cross.y -= stride.y ;
				
		}else{
        // if next == 2 
				
			// Compute weight.
			int indexX = Source.x + __float2int_rn( 0.5f*(crossOld+cross.z)*fVec.x ) ;
			int indexY = Source.y + __float2int_rn( 0.5f*(crossOld+cross.z)*fVec.y ) ;
			int indexZ = Source.z + __float2int_rn( 0.5f*(crossOld+cross.z)*fVec.z ) ;
			Weight += Slowness[ (indexZ*shared_Y + indexY)*shared_X + indexX ]*(crossOld-cross.z) ;

			// For next crossing point.
			crossOld = cross.z ;
			--nVec.z ;
			cross.z -= stride.z ;
				
		}
        }
			
	}
	
	// Weight at source
	Weight += Slowness[ (Source.z*shared_Y + Source.y)*shared_X + Source.x]*crossOld ;
    Weight *= sqrt( powf(__int2float_rn(ix), 2) + powf(__int2float_rn(iy), 2) + powf(__int2float_rn(iz), 2) ) ;

return Weight ;}

/// Calculate new traveltime 
__device__ float FIM(float Tx, float Ty, float Tz, const float S)
{     
    float TT = 3.0f*powf(S,2) -powf(Tx-Ty,2)-powf(Tx-Tz,2)-powf(Ty-Tz,2) ;
    if( TT < 0 )
    {
        // Let Tx = Largest (use TT as temp)
        if(Tx < Ty){ TT = Tx ; Tx = Ty ; Ty = TT ; } 
	    if(Tx < Tz){ TT = Tx ; Tx = Tz ; Tz = TT ; }	
	    TT = 0.5f*( Ty+Tz + sqrt(2.0f*powf(S,2) - powf(Ty-Tz,2)) ) ; 
	    
    }else{
 
        TT = ( Tx+Ty+Tz + sqrt(TT) )/3.0f ; 
 
    }
    
    	   
return TT ; } // If it is NaN, then newTt<Tt is false anyway.

// Greatest Common Divisor for edge reduction
__device__ int GCD(int a, int b)
{
    a = abs(a) ;
    b = abs(b) ;
    int c ;
    while( a!=0 )
    {
        c = a ;
        a = b%a ;
        b = c ;
    }

return b; }

__device__ void checkTraveltime(const float *Traveltime, float *Tt, int *rayPath, const int3 posWorker, int i, int j, int k, float weight)
{
    float newTt = weight + Traveltime[ posWorker.x+i +shared_X*(posWorker.y+j +shared_Y*(posWorker.z+k)) ] ; 
    if( newTt<*Tt ){
		*Tt = newTt ;
		*rayPath = ((radius_Z+k)*group_Y + radius_Y+j)*group_X + radius_X+i ;
	}

}

//**************** AB functions ****************//

// Upload Slowness x4 and Traveltime x4 to shared memory
__device__ void uploadShared(float *Slowness, float *Traveltime)
{
	float temp ;
	int3  Stride = make_int3( 2*radius_X, 2*radius_Y, 2*radius_Z ) ;
        
	// (1,1,1)
	int3    ptrMap = make_int3( sizeof(float)*(blockIdx.x*blockDim.x + threadIdx.x), blockIdx.y*blockDim.y + threadIdx.y,  blockIdx.z*blockDim.z + threadIdx.z);
	int3 ptrShared = make_int3( threadIdx.x, threadIdx.y, threadIdx.z ) ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	// (2,1,1)
       ptrMap.x += sizeof(float)*Stride.x ;
	ptrShared.x += Stride.x ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	 // (2,2,1)
	   ptrMap.y += Stride.y ;
	ptrShared.y += Stride.y ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	 // (1,2,1)
       ptrMap.x -= sizeof(float)*Stride.x ;
	ptrShared.x -= Stride.x ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	//********************************************************************
	
	// (1,2,2)
	   ptrMap.z += Stride.z ;
	ptrShared.z += Stride.z ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	// (2,2,2)
       ptrMap.x += sizeof(float)*Stride.x ;
	ptrShared.x += Stride.x ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	 // (2,1,2)
	   ptrMap.y -= Stride.y ;
	ptrShared.y -= Stride.y ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;

	 // (1,1,2)
       ptrMap.x -= sizeof(float)*Stride.x ;
	ptrShared.x -= Stride.x ;
	surf3Dread(&temp,SlownessMap, ptrMap.x, ptrMap.y, ptrMap.z );
	Slowness[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;
	surf3Dread(&temp,Traveltime_A, ptrMap.x, ptrMap.y, ptrMap.z);
	Traveltime[ (ptrShared.z*shared_Y + ptrShared.y)*shared_X + ptrShared.x ] = temp ;


}


/// Compare New and Old traveltime
__device__ void compareTraveltime(const float *Traveltime, const float *Slowness, bool *isUpdated)
{
	int3 posWorker =  make_int3( threadIdx.x+radius_X, threadIdx.y+radius_Y,  threadIdx.z+radius_Z) ; // posWorker on shMem ###
	float Tt = Traveltime[ (posWorker.z*shared_Y + posWorker.y)*shared_X + posWorker.x ] ;
	int rayPath = -100 ;
	
	
	// Start:: Comparing neighbor != (0,0,0) ***************************************
    
	//-----------------------------------------------------------------------
	float  T1, T2,  T3,  T4, T5, Tt1 ;
	float Tz1, Tz2, Tz3, Tz4 ;
	float Tz5, Tz6, Tz7, Tz8 ;
	float Te1, Te2, Te3 ;
	
    // As 2D, START:: Level z = -2 ------------------------------------------

          Tt1 = SPR(Slowness,-2,-1,-2, posWorker); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-1,-2, Tt1); 

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-1,-2, SPR(Slowness,-1,-1,-2, posWorker));	

	      T1  = SPR(Slowness,-1,-2,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-2,-2,  T1);	

	      T5  = SPR(Slowness,-2,-2,-1, posWorker); 
	      Tz1 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,-1,  T5);

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,-2, FIM(T1,Tt1,T5, Slowness[posWorker.x-2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z-2 )) ] ));
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,-2, SPR(Slowness,-2,-2,-2, posWorker));
	
	// -----------------
	
	      T2  = SPR(Slowness,0,-1,-2, posWorker);
	      Te1 = T2 ;  
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-1,-2,  T2);

	      T3  = SPR(Slowness,1,-2,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-2,-2,  T3);

	      T5  = SPR(Slowness,0,-2,-1, posWorker);
	      Tz5 = T5 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,-1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,-2, FIM(min(T1,T3),T2,T5 , Slowness[posWorker.x + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z-2 )) ] ));
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,-2, SPR(Slowness,0,-2,-2, posWorker));

    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-1,-2, SPR(Slowness,1,-1,-2, posWorker));
	
	      T1  = SPR(Slowness,2,-1,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-1,-2,  T1);
	
	      T5  = SPR(Slowness,2,-2,-1, posWorker); 
	      Tz2 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,-1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,-2, FIM(T3,T1,T5 , Slowness[posWorker.x+2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,-2, SPR(Slowness,2,-2,-2, posWorker));
	
	// -----------------
	 
	      T2  = SPR(Slowness,1,0,-2, posWorker); 
	      Te2 = T2 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,0,-2,  T2);
	
	      T3  = SPR(Slowness,2,1,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,1,-2,  T3);
	
	      T5  = SPR(Slowness,2,0,-1, posWorker); 
	      Tz6 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,-1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,-2, FIM(T2,min(T1,T3),T5 , Slowness[posWorker.x+2 + shared_X*( posWorker.y +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,-2, SPR(Slowness,2,0,-2, posWorker));
 
    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,1,-2, SPR(Slowness,1,1,-2, posWorker));
	
	      T1  = SPR(Slowness,1,2,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,2,-2,  T1);
	
	      T5  = SPR(Slowness,2,2,-1, posWorker);
	      Tz3 = T5 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,-1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,-2, FIM(T1,T3,T5 , Slowness[posWorker.x+2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,-2, SPR(Slowness,2,2,-2, posWorker));
   
    // -----------------
   
	      T2  = SPR(Slowness, 0,1,-2, posWorker);
	      Te3 = T2 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,1,-2, T2);
	
	      T3  = SPR(Slowness,-1,2,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,2,-2, T3);
	
          T5  = SPR(Slowness, 0,2,-1, posWorker); 
          Tz7 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,-1, T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,-2, FIM(min(T1,T3),T2,T5 , Slowness[posWorker.x + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,-2, SPR(Slowness,0,2,-2, posWorker));
   
    // -----------------
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,1,-2, SPR(Slowness,-1,1,-2, posWorker));
	
	      T1  = SPR(Slowness,-2,1,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,1,-2, T1);
	
	      T5  = SPR(Slowness,-2,2,-1, posWorker); 
	      Tz4 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,-1, T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,-2, FIM(T3,T1,T5 , Slowness[posWorker.x-2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,-2, SPR(Slowness,-2,2,-2, posWorker));
	
	// -----------------
	
	      T2  = SPR(Slowness,-1,0,-2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,0,-2, T2);
	
	      T5  = SPR(Slowness,-2,0,-1, posWorker); 
	      Tz8 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,-1, T5);
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,-2, FIM(T2,min(T1,Tt1),T5 , Slowness[posWorker.x-2 + shared_X*( posWorker.y +shared_Y*( posWorker.z-2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,-2, SPR(Slowness,-2,0,-2, posWorker));
    
    // -----------------
    // Special node (0,0)
    
    	  T4  = SPR(Slowness,0,0,-1, posWorker);
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,-1, T4);
    
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,-2, FIM(min(T2,Te2),min(Te1,Te3), T4, Slowness[posWorker.x + shared_X*( posWorker.y +shared_Y*( posWorker.z-2 )) ] )); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,-2, SPR(Slowness,0,0,-2, posWorker));

    // As 2D,   END:: Level z = -2 ------------------------------------------
   	// As 2D, START:: Level z = -1 ------------------------------------------

    // Comparing neighbor (With edge reduction)
	// For node (i,j,-1) except (0,0,-1) 
	for(int j = -radius_Y ; j<=radius_Y ; ++j )
	{
		for(int i = -radius_X ; i <= radius_X ; ++i )
		{
		    if( GCD(i,j)==1 )
		    {
                checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,i,j,-1, SPR(Slowness,i,j,-1, posWorker));
		    }
		}
	}
 
    // As 2D,   END:: Level z = -1 ------------------------------------------
    // As 2D, START:: Level z =  0 ------------------------------------------
 
 
          Tt1 = SPR(Slowness,-2,-1,0, posWorker); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-1,0, Tt1); 

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-1,0, SPR(Slowness,-1,-1,0, posWorker));	

	      T1  = SPR(Slowness,-1,-2,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-2,0,  T1);	

          T2  = Tz1 ;
	      T5  = SPR(Slowness,-2,-2,1, posWorker); 
	      Tz1 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,1,  T5);

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,0, FIM(T1,Tt1,min(T2,T5), Slowness[posWorker.x-2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z )) ] ));
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,0, SPR(Slowness,-2,-2,0, posWorker));
	
	// -----------------
	
	      T3  = SPR(Slowness,0,-1,0, posWorker);
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-1,0,  T3);

	      T4  = SPR(Slowness,1,-2,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-2,0,  T4 );

          T2  = Tz5 ; 
	      T5  = SPR(Slowness,0,-2,1, posWorker);
	      Tz5 = T5 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,1,  T5);
	
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,0, FIM(min(T1,T4),T3,min(T2,T5), Slowness[posWorker.x + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z )) ] )); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,0, SPR(Slowness,0,-2,0, posWorker));

    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-1,0, SPR(Slowness,1,-1,0, posWorker));
	
	      T1  = SPR(Slowness,2,-1,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-1,0,  T1);
	
	      T2  = Tz2 ;
	      T5  = SPR(Slowness,2,-2,1, posWorker);
	      Tz2 = T5 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,0, FIM(T4,T1,min(T2,T5) , Slowness[posWorker.x+2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,0, SPR(Slowness,2,-2,0, posWorker));
	
	// -----------------
	     
	      T3  = SPR(Slowness,1,0,0, posWorker);
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,0,0,  T3);
	
	      T4  = SPR(Slowness,2,1,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,1,0,  T4);
 
          T2  = Tz6 ; 
	      T5  = SPR(Slowness,2,0,1, posWorker); 
	      Tz6 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,1,  T5);
 
  	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,0, FIM(T3,min(T1,T4),min(T2,T5), Slowness[posWorker.x+2 + shared_X*( posWorker.y +shared_Y*( posWorker.z )) ] )); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,0, SPR(Slowness,2,0,0, posWorker));
 
    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,1,0, SPR(Slowness,1,1,0, posWorker));
	
	      T1  = SPR(Slowness,1,2,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,2,0,  T1);
	
	      T2  = Tz3 ;
	      T5  = SPR(Slowness,2,2,1, posWorker); 
	      Tz3 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,1,  T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,0, FIM(T1,T4,min(T2,T5) , Slowness[posWorker.x+2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,0, SPR(Slowness,2,2,0, posWorker));
   
    // -----------------
   
          T3  = SPR(Slowness, 0,1,0, posWorker);
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,1,0, T3);
	
	      T4  = SPR(Slowness,-1,2,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,2,0, T4);
	
	      T2  = Tz7 ; 
	      T5  = SPR(Slowness, 0,2,1, posWorker); 
	      Tz7 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,1,  T5);
 
 	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,0, FIM(min(T1,T4),T3,min(T2,T5), Slowness[posWorker.x + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z )) ] )); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,0, SPR(Slowness,0,2,0, posWorker));
   
    // -----------------
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,1,0, SPR(Slowness,-1,1,0, posWorker));
	
	      T1  = SPR(Slowness,-2,1,0, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,1,0, T1);
	
	      T2  = Tz4 ; 
	      T5  = SPR(Slowness,-2,2,1, posWorker); 
	      Tz4 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,1, T5);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,0, FIM(T4,T1,min(T2,T5) , Slowness[posWorker.x-2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,0, SPR(Slowness,-2,2,0, posWorker));
	
	// -----------------
	
	      T3  = SPR(Slowness,-1,0,0, posWorker);
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,0,0, T3);
	 
	      T2  = Tz8 ; 
	      T5  = SPR(Slowness,-2,0,1, posWorker); 
	      Tz8 = T5 ;
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,1,  T5);
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,0, FIM(T3,min(Tt1, T1),min(T2,T5), Slowness[posWorker.x-2 + shared_X*( posWorker.y +shared_Y*( posWorker.z )) ] )); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,0, SPR(Slowness,-2,0,0, posWorker));
    
 
    // As 2D,   END:: Level z =  0 ------------------------------------------
    // As 2D, START:: Level z = +1 ------------------------------------------
    
    // Comparing neighbor (With edge reduction)
	// For node (i,j,1) except (0,0,1) 
	for(int j = -radius_Y ; j<=radius_Y ; ++j )
	{
		for(int i = -radius_X ; i <= radius_X ; ++i )
		{
		    if( GCD(i,j)==1 )
		    {
                checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,i,j,1, SPR(Slowness,i,j,1, posWorker));
		    }
		}
	}
    
    // As 2D,   END:: Level z = +1 ------------------------------------------
    // As 2D, START:: Level z = +2 ------------------------------------------

          Tt1 = SPR(Slowness,-2,-1,2, posWorker); 
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-1,2, Tt1); 

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-1,2, SPR(Slowness,-1,-1,2, posWorker));	

	      T1  = SPR(Slowness,-1,-2,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,-2,2,  T1);	

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,2, FIM(T1,Tt1,Tz1, Slowness[posWorker.x-2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z+2 )) ] ));
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,-2,2, SPR(Slowness,-2,-2,2, posWorker));
	
	// -----------------
	
	      T2  = SPR(Slowness,0,-1,2, posWorker); 
	      Te1 = T2 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-1,2,  T2);

	      T3  = SPR(Slowness,1,-2,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-2,2,  T3);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,2, FIM(min(T1,T3),T2,Tz5 , Slowness[posWorker.x + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z+2 )) ] ));
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,-2,2, SPR(Slowness,0,-2,2, posWorker));

    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,-1,2, SPR(Slowness,1,-1,2, posWorker));
	
	      T1  = SPR(Slowness,2,-1,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-1,2,  T1);
	
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,2, FIM(T3,T1,Tz2 , Slowness[posWorker.x+2 + shared_X*( posWorker.y-2 +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,-2,2, SPR(Slowness,2,-2,2, posWorker));
	
	// -----------------
	 
	      T2  = SPR(Slowness,1,0,2, posWorker); 
	      Te2 = T2 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,0,2,  T2);
	
	      T3  = SPR(Slowness,2,1,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,1,2,  T3);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,2, FIM(T2,min(T1,T3),Tz6 , Slowness[posWorker.x+2 + shared_X*( posWorker.y +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,0,2, SPR(Slowness,2,0,2, posWorker));
 
    // -----------------

	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,1,2, SPR(Slowness,1,1,2, posWorker));
	
	      T1  = SPR(Slowness,1,2,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,1,2,2,  T1);
	

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,2, FIM(T1,T3,Tz3 , Slowness[posWorker.x+2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,2,2,2, SPR(Slowness,2,2,2, posWorker));
   
    // -----------------
   
	      T2  = SPR(Slowness, 0,1,2, posWorker); 
	      Te3 = T2 ; 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,1,2, T2);
	
	      T3  = SPR(Slowness,-1,2,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,2,2, T3);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,2, FIM(min(T1,T3),T2,Tz7 , Slowness[posWorker.x + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker, 0,2,2, SPR(Slowness,0,2,2, posWorker));
   
    // -----------------
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,1,2, SPR(Slowness,-1,1,2, posWorker));
	
	      T1  = SPR(Slowness,-2,1,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,1,2, T1);

    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,2, FIM(T3,T1,Tz4 , Slowness[posWorker.x-2 + shared_X*( posWorker.y+2 +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,2,2, SPR(Slowness,-2,2,2, posWorker));
	
	// -----------------
	
	      T2  = SPR(Slowness,-1,0,2, posWorker); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-1,0,2, T2);
	
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,2, FIM(T2,min(T1,Tt1),Tz8 , Slowness[posWorker.x-2 + shared_X*( posWorker.y +shared_Y*( posWorker.z+2 )) ] ));   
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,-2,0,2, SPR(Slowness,-2,0,2, posWorker));
    
    // -----------------
    // Special node (0,0)	
   
          T4  = SPR(Slowness,0,0,1, posWorker);
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,1, T4);
    	
    checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,2, FIM(min(T2,Te2),min(Te1,Te3),T4 , Slowness[posWorker.x + shared_X*( posWorker.y +shared_Y*( posWorker.z+2 )) ] )); 
	checkTraveltime(Traveltime,&Tt,&rayPath,posWorker,0,0,2, SPR(Slowness,0,0,2, posWorker));


    // As 2D, END:: Level z = +2 --------------------------------------------
 
	//-----------------------------------------------------------------------
	
    // End:: Comparing neighbor != (0,0,0) ***************************************

	// Write outTraveltime
    int GlobalID_X = blockIdx.x*blockDim.x + threadIdx.x ;
    int GlobalID_Y = blockIdx.y*blockDim.y + threadIdx.y ;
	int GlobalID_Z = blockIdx.z*blockDim.z + threadIdx.z ;
    surf3Dwrite(Tt, Traveltime_B, sizeof(float)*(GlobalID_X+radius_X), GlobalID_Y+radius_Y,  GlobalID_Z+radius_Z);
		
	// if threadUpdate -> isUpdated[0] = true ;
	if( rayPath != -100 ){
      *isUpdated = true ;
	  surf3Dwrite(rayPath, RaypathMap, sizeof(int)*GlobalID_X, GlobalID_Y, GlobalID_Z);
    }

}

//**************** Global CUDA ****************//

__global__ void cuda_PSPS_Solver(int *Running)
{
	
	// Define shared variables (1)
	__shared__ bool isAnyUpdated ;
	__shared__ bool needUpdate  ;


	/// Need update ?
	if( threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 ){
        needUpdate   = false ;
        isAnyUpdated = false ;
	}
	__syncthreads();
	if( threadIdx.x<3 && threadIdx.y<3 && threadIdx.z<3 ){
		int data ; 
		surf3Dread(&data, UpdateMap_A, sizeof(int)*(blockIdx.x+threadIdx.x), blockIdx.y+threadIdx.y, blockIdx.z+threadIdx.z);
	    if( data==1 ){
	        needUpdate = true ;
	    }
	}
	__syncthreads();
	if( !needUpdate ){ return; }



    // Define shared variables (2)
	extern __shared__ float sharedVar[] ;
	float *Slowness   = &sharedVar[0] ;  
	float *Traveltime = &sharedVar[ shared_X*shared_Y*shared_Z ] ; 
    //isAnyUpdated = false ; //%%%

	/// Upload Slowness x4 and Traveltime x4 to shared memory	
    uploadShared( Slowness, Traveltime); 
	__syncthreads();

	/// Compare New and Old traveltime
	compareTraveltime(Traveltime, Slowness, &isAnyUpdated);
	__syncthreads();

/*
    // For no update map scheme  
    // ### don't forget to set isUpdate[0] = false and remove the __syncthreads() 
    if( isAnyUpdated ){    
//	    ++(*Running) ;
        atomicAdd(Running,1);
    }
*/


	/// IsUpdate ? -> update running and UpdateMap.
	if( threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 )
	{
        if( isAnyUpdated ){
            ++(*Running) ;
            //atomicAdd(Running,1);
        }
	    surf3Dwrite( (isAnyUpdated) ? 1:0 , UpdateMap_B, sizeof(int)*(blockIdx.x+1), blockIdx.y+1, blockIdx.z+1);
	}


}


//**************** End CUDA ****************//

bool PSPS_CUDA_ExtendedBF_3D_Solver::Compute()
{
	/// Create Compute Shader (Solver) ///
   // Compute some useful numbers.
   unsigned int shared[DIM] ;
   for(unsigned int i=0 ; i<DIM ; ++i)
   {
      length[i] = res[i]+block[i] ;
      shared[i] = 2*block[i] ;
   }
   side = block[0]/2 ;
   front = length[0]*block[1]/2 ;
   top = length[0]*length[1]*block[2]/2 ;
   
    //******************************************************************************************
    
   /// Add dummy vertices ONLY 3D ///
   addDummyVertices();

	// Set constant values
	int tempSh[] = { (int)shared[0], (int)shared[1], (int)shared[2]} ;
	hipMemcpyToSymbol(HIP_SYMBOL(shared_X), &tempSh[0], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(shared_Y), &tempSh[1], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(shared_Z), &tempSh[2], sizeof(int));
	
	int tempGr[] = { (int)block[0]+1, (int)block[1]+1, (int)block[2]+1} ;
	hipMemcpyToSymbol(HIP_SYMBOL(group_X), &tempGr[0], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(group_Y), &tempGr[1], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(group_Z), &tempGr[2], sizeof(int));
	
	int tempRa[] = {(int)side, (int)block[1]/2, (int)block[2]/2} ;
	hipMemcpyToSymbol(HIP_SYMBOL(radius_X), &tempRa[0], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(radius_Y), &tempRa[1], sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(radius_Z), &tempRa[2], sizeof(int));
	
	// Set Maps
   hipChannelFormatDesc channelDescFloat = hipCreateChannelDesc(32,0,0,0, hipChannelFormatKindFloat); 
   hipChannelFormatDesc channelDescInt   = hipCreateChannelDesc(32,0,0,0, hipChannelFormatKindSigned); 
	   
   // Slowness   
   hipArray* cuArraySlowness ;
   hipMalloc3DArray(&cuArraySlowness, &channelDescFloat, make_hipExtent(length[0], length[1], length[2]), hipArraySurfaceLoadStore);
   hipMemcpy3DParms paramInS = {0} ;
	  paramInS.srcPtr = make_hipPitchedPtr(&slowness[0], length[0]*sizeof(float), length[0], length[1]);
	  paramInS.dstArray = cuArraySlowness ;
      paramInS.kind =  hipMemcpyHostToDevice ;
      paramInS.extent = make_hipExtent( length[0], length[1], length[2]) ;
   hipMemcpy3D(&paramInS);
   cudaBindSurfaceToArray(SlownessMap, cuArraySlowness);
   // Raypath 
   hipArray* cuArrayRaypath ;
   hipMalloc3DArray(&cuArrayRaypath, &channelDescInt, make_hipExtent(res[0], res[1], res[2]), hipArraySurfaceLoadStore);
   hipMemcpy3DParms paramInR = {0} ;
	  paramInR.srcPtr = make_hipPitchedPtr(&raypath[0], res[0]*sizeof(int), res[0], res[1]) ;
	  paramInR.dstArray = cuArrayRaypath ;
      paramInR.kind =  hipMemcpyHostToDevice ;
      paramInR.extent = make_hipExtent( res[0], res[1], res[2]) ;
   hipMemcpy3D(&paramInR);
   cudaBindSurfaceToArray(RaypathMap, cuArrayRaypath);
   // Traveltime x2
   hipArray *cuArrayTraveltime_A, *cuArrayTraveltime_B ;
   hipMalloc3DArray(&cuArrayTraveltime_A, &channelDescFloat, make_hipExtent( length[0], length[1], length[2]), hipArraySurfaceLoadStore);
   hipMalloc3DArray(&cuArrayTraveltime_B, &channelDescFloat, make_hipExtent( length[0], length[1], length[2]), hipArraySurfaceLoadStore);
   hipMemcpy3DParms paramInT = {0} ;
	  paramInT.srcPtr = make_hipPitchedPtr(&traveltime[0], length[0]*sizeof(float), length[0], length[1]) ;
	  paramInT.dstArray = cuArrayTraveltime_A ;
      paramInT.kind =  hipMemcpyHostToDevice ;
      paramInT.extent = make_hipExtent( length[0], length[1], length[2]) ;
   hipMemcpy3D(&paramInT);
   	  paramInT.dstArray = cuArrayTraveltime_B ;
   hipMemcpy3D(&paramInT);
   //cudaBindSurfaceToArray(Traveltime_A, cuArrayTraveltime_A);
   //cudaBindSurfaceToArray(Traveltime_B, cuArrayTraveltime_B);

   // UpdateMap x2
   hipArray *cuArrayUpdateMap_A, *cuArrayUpdateMap_B ;
   hipMalloc3DArray(&cuArrayUpdateMap_A, &channelDescInt, make_hipExtent( nWorkGroup[0]+2, nWorkGroup[1]+2, nWorkGroup[2]+2), hipArraySurfaceLoadStore);
   hipMalloc3DArray(&cuArrayUpdateMap_B, &channelDescInt, make_hipExtent( nWorkGroup[0]+2, nWorkGroup[1]+2, nWorkGroup[2]+2), hipArraySurfaceLoadStore);
   hipMemcpy3DParms paramInU = {0} ;
	  paramInU.srcPtr = make_hipPitchedPtr(&updateMap[0], (nWorkGroup[0]+2)*sizeof(int), (nWorkGroup[0]+2), (nWorkGroup[1]+2)) ;
	  paramInU.dstArray = cuArrayUpdateMap_A ;
      paramInU.kind =  hipMemcpyHostToDevice ;
      paramInU.extent = make_hipExtent( nWorkGroup[0]+2, nWorkGroup[1]+2, nWorkGroup[2]+2) ;
   hipMemcpy3D(&paramInU);
   //cudaBindSurfaceToArray(UpdateMap_A, cuArrayUpdateMap_A);
   //cudaBindSurfaceToArray(UpdateMap_B, cuArrayUpdateMap_B);

   //******************************************************************************************

    // Set status
   int running = 0 ; 
   int *cuRunning ;
   hipMalloc(&cuRunning, sizeof(int));
   hipMemcpy(cuRunning, &running, sizeof(int), hipMemcpyHostToDevice);
   bool AB = true ;

   { ///***** =.\= Parallel Shortest Path Solver (CUDA) =.\= *****///

      ///*** Main loop ***///
      int loop ; 
	  dim3 dimBlock( block[0], block[1], block[2] ); 
      dim3 dimGroup( nWorkGroup[0], nWorkGroup[1], nWorkGroup[2] );
	  size_t nSharedVar = 2*shared[0]*shared[1]*shared[2]*sizeof(float) ;

	  // Current Time
      time_t rawtime ;
      
      // Start
      std::cout << "\nPSPS_CUDA_ExtendedBF >> Start \'Extended\' Bellman-Ford (CUDA).\n" ;
      time(&rawtime);
      std::cout << "   START:: " << ctime(&rawtime) ;
      
      // Start timer
      auto start_time = std::chrono::high_resolution_clock::now();
	  
	  // swap==0  A-> B
      // swap==1  B-> A
	  for(loop=0 ; loop<maxLoop ; ++loop)
      {		   
         /// AB ///
		cudaBindSurfaceToArray(Traveltime_A, cuArrayTraveltime_A, channelDescFloat);
        cudaBindSurfaceToArray(Traveltime_B, cuArrayTraveltime_B, channelDescFloat);
        cudaBindSurfaceToArray(UpdateMap_A, cuArrayUpdateMap_A, channelDescInt);
        cudaBindSurfaceToArray(UpdateMap_B, cuArrayUpdateMap_B, channelDescInt);
         /***( Solving =/.= )***/
		 cuda_PSPS_Solver<<<dimGroup,dimBlock,nSharedVar>>>(cuRunning);

		 // Read status 
		 hipMemcpy(&running, cuRunning, sizeof(int), hipMemcpyDeviceToHost);
         //std::cout << "  Loop = " << loop << "  Running = " << running << " \n" ; 
         if( running==0 ){ isFinish = true ; AB = false ; break; }
         running = 0 ; 
		 hipMemcpy(cuRunning, &running, sizeof(int), hipMemcpyHostToDevice);
         
         ++loop;
         
         /// BA ///
		cudaBindSurfaceToArray(Traveltime_A, cuArrayTraveltime_B, channelDescFloat);
        cudaBindSurfaceToArray(Traveltime_B, cuArrayTraveltime_A, channelDescFloat);
        cudaBindSurfaceToArray(UpdateMap_A, cuArrayUpdateMap_B, channelDescInt);
        cudaBindSurfaceToArray(UpdateMap_B, cuArrayUpdateMap_A, channelDescInt);
         /***( Solving =/.= )***/
		 cuda_PSPS_Solver<<<dimGroup,dimBlock,nSharedVar>>>(cuRunning);

		 // Read status 
		 hipMemcpy(&running, cuRunning, sizeof(int), hipMemcpyDeviceToHost);
         //std::cout << "  Loop = " << loop << "  Running = " << running << " \n" ; 
         if( running==0 ){ isFinish = true ; AB = true ; break; }
         running = 0 ; 
		 hipMemcpy(cuRunning, &running, sizeof(int), hipMemcpyHostToDevice);
        
      }

      // Stop timer
      auto end_time = std::chrono::high_resolution_clock::now();
      
      // Finish
      time(&rawtime);
      std::cout << "  FINISH:: " << ctime(&rawtime) ;
      std::cout << "PSPS_CUDA_ExtendedBF >> Finish \'Extended\' Bellman-Ford (CUDA).\n\n" ;
        
      std::cout << " *** Runtime = " << std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time).count() << " seconds. ***\n";
            
      // Save nloop
      nloop += loop ;
      // Display now state
      if( isFinish ){
         std::cout << " *** Shortest paths are all found, after total " << nloop << " iterations. ***\n\n" ;
      }else{
         std::cout << " *** Solver is not finish finding Shortest paths, after total " << nloop << " iterations. ***\n\n" ;
      }
   }
   // Clear status
   hipFree(cuRunning);   
   
   //******************************************************************************************

   /// Retrieve Maps ///
	// Traveltime
    hipMemcpy3DParms paramOutT = {0} ;
	  if( !AB ){
         paramOutT.srcArray = cuArrayTraveltime_B ; 
      }else{
		 paramOutT.srcArray = cuArrayTraveltime_A ;
	  }
	  paramOutT.dstPtr = make_hipPitchedPtr(&traveltime[0], length[0]*sizeof(float), length[0], length[1]) ;
	  paramOutT.kind   = hipMemcpyDeviceToHost ;
      paramOutT.extent = make_hipExtent( length[0], length[1], length[2]) ;
    hipMemcpy3D(&paramOutT);

	// Raypath
	hipMemcpy3DParms paramOutR = {0} ;
	  paramOutR.srcArray = cuArrayRaypath ;
	  paramOutR.dstPtr = make_hipPitchedPtr(&raypath[0], res[0]*sizeof(int), res[0], res[1]) ;
      paramOutR.kind =  hipMemcpyDeviceToHost ;
      paramOutR.extent = make_hipExtent( res[0], res[1], res[2]) ;
    hipMemcpy3D(&paramOutR);
	
	// UpdateMap
	if( !isFinish )
	{
		 hipMemcpy3DParms paramOutU = {0} ;
			if( !AB ){
				paramOutU.srcArray = cuArrayUpdateMap_B ;
			}else{
				paramOutU.srcArray = cuArrayUpdateMap_A ;
			}
			paramOutU.dstPtr = make_hipPitchedPtr(&updateMap[0], (nWorkGroup[0]+2)*sizeof(int), (nWorkGroup[0]+2), (nWorkGroup[1]+2)) ;
			paramOutU.kind =  hipMemcpyDeviceToHost ;
			paramOutU.extent = make_hipExtent( nWorkGroup[0]+2, nWorkGroup[1]+2, nWorkGroup[2]+2) ;
		hipMemcpy3D(&paramOutU);
	}

   /// Remove dummy vertices ONLY 3D ///
   removeDummyVertices();

   /// Clear Shader and Buffers ///
   hipFreeArray(cuArraySlowness); 
   hipFreeArray(cuArrayRaypath);
   hipFreeArray(cuArrayTraveltime_A);
   hipFreeArray(cuArrayTraveltime_B);
   hipFreeArray(cuArrayUpdateMap_A);
   hipFreeArray(cuArrayUpdateMap_B);

return true; }

/************************************* Private *************************************/

void PSPS_CUDA_ExtendedBF_3D_Solver::addDummyVertices()
{
   unsigned int at ;
   bool isNegative = false ;
   std::vector<TyPe> temp ;

   /// Slowness map.
   // Copy data
   temp.assign( slowness.begin(), slowness.end() ) ;
   // Reinput the data
   slowness.assign( length[0]*length[1]*length[2], 1.0f/0.0f) ;
   at = top+front+side ;
   for(unsigned int k=0 ; k<res[2] ; ++k)
   {
		for(unsigned int j=0 ; j<res[1] ; ++j)
		{
			for(unsigned int i=0 ; i<res[0] ; ++i)
			{
				if( temp[ (k*res[1]+j)*res[0]+i ] < 0 ){ isNegative = true ; } // Slowness must only be positive ? ###
				slowness[ at+i ] = std::abs( temp[ (k*res[1]+j)*res[0]+i ]*stride[0] ) ;
			}
			at += res[0]+2*side ;
		}
		at += 2*front ; 
   }
   if( isNegative ){
      std::cout << "Warning !! Negative slowness is detected -> using absolute value.\n" ;
   }

   /// Traveltime map.
   // Copy data
   temp.assign( traveltime.begin(), traveltime.end() ) ;
   // Reinput the data
   traveltime.assign( length[0]*length[1]*length[2], 1.0f/0.0f) ;
   at = top+front+side ;
   for(unsigned int k=0 ; k<res[2] ; ++k)
   {
		for(unsigned int j=0 ; j<res[1] ; ++j)
		{
			for(unsigned int i=0 ; i<res[0] ; ++i)
			{
				traveltime[ at+i ] = temp[ (k*res[1]+j)*res[0]+i ] ;
			}
			at += res[0]+2*side ;
		}
		at += 2*front ; 
   }

   /// UpdateMap.
   // Copy data
   std::vector<int> tem ;
   tem.assign( updateMap.begin(), updateMap.end() ) ;
   // Reinput the data
   updateMap.assign( (nWorkGroup[0]+2)*(nWorkGroup[1]+2)*(nWorkGroup[2]+2), 0) ;
   at = (nWorkGroup[0]+2)*(nWorkGroup[1]+2)+(nWorkGroup[0]+2)+1 ;
   for(unsigned int k=0 ; k<nWorkGroup[2] ; ++k)
   {
		for(unsigned int j=0 ; j<nWorkGroup[1] ; ++j)
		{
			for(unsigned int i=0 ; i<nWorkGroup[0] ; ++i)
			{
				updateMap[ at+i ] = tem[ (k*nWorkGroup[1]+j)*nWorkGroup[0]+i ] ;
			}
			at += nWorkGroup[0]+2 ;
		}
		at += 2*(nWorkGroup[0]+2) ;
   }

return; }

void PSPS_CUDA_ExtendedBF_3D_Solver::removeDummyVertices()
{
   unsigned int at ;
   std::vector<TyPe> temp ;

   /// Slowness map
   // Copy data
   temp.assign( slowness.begin(), slowness.end() ) ;
   // Reinput the data
   slowness.resize( res[0]*res[1]*res[2] ) ;
   at = top+front+side ;
   for(unsigned int k=0 ; k<res[2] ; ++k)
   {
		for(unsigned int j=0 ; j<res[1] ; ++j)
		{
			for(unsigned int i=0 ; i<res[0] ; ++i)
			{
				slowness[ (k*res[1]+j)*res[0]+i ] = temp[ at+i ]/stride[0] ;
			}
			at += res[0]+2*side ;
		}
		at += 2*front ; 
   }
   
   /// Traveltime map
   // Copy data
   temp.assign( traveltime.begin(), traveltime.end() ) ;
   // Reinput the data
   traveltime.resize( res[0]*res[1]*res[2] ) ;
   at = top+front+side ;
   for(unsigned int k=0 ; k<res[2] ; ++k)
   {
		for(unsigned int j=0 ; j<res[1] ; ++j)
		{
			for(unsigned int i=0 ; i<res[0] ; ++i)
			{
				traveltime[ (k*res[1]+j)*res[0]+i ] = temp[ at+i ] ;
			}
			at += res[0]+2*side ;
		}
		at += 2*front ; 
   }
   
   /// UpdateMap.
   if( isFinish ){ return ; } //No more update map
   // Copy data
   std::vector<int> tem ;
   tem.assign( updateMap.begin(), updateMap.end() ) ;
   // Reinput the data
   updateMap.resize( nWorkGroup[0]*nWorkGroup[1]*nWorkGroup[2] ) ;
   at = (nWorkGroup[0]+2)*(nWorkGroup[1]+2)+(nWorkGroup[0]+2)+1 ;
   for(unsigned int k=0 ; k<nWorkGroup[2] ; ++k)
   {
		for(unsigned int j=0 ; j<nWorkGroup[1] ; ++j)
		{
			for(unsigned int i=0 ; i<nWorkGroup[0] ; ++i)
			{
				updateMap[ (k*nWorkGroup[1]+j)*nWorkGroup[0]+i ] = tem[ at+i ] ;
			}
			at += nWorkGroup[0]+2 ;
		}
		at += 2*(nWorkGroup[0]+2) ;
   }
   

return; }
